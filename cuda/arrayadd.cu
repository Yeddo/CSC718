
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void addArray(int *a, int *b, int*c, int length)
{
    int tid = blockIdx.x;
    if(tid < length)
        c[tid] = a[tid] + b[tid];


}
void debug(int * array, int length)
{
    int i;
    for( i = 0; i < length; i++)
    {
        printf("%2d: %6d  ", i, array[i]);
    }
    printf("\n");
    return;
}

void printArray(int * array, int length)
{
    int i;
    for( i = 0; i < length; i++)
    {
        printf("%d: %6d\n", i, array[i]);
    }
    return;
}

void generateArray(int **array, int length)
{
    int i;
    *array = (int*)malloc(sizeof(int) * length);
    memset ((void *) *array, 0, sizeof(int) * length);   
    printf("I got here\n");
    for(i = 0; i < length; i++)
    {
        (*array)[i] = rand() % 1000;
    }
    printf("I filled it up\n");
   // debug(*array, length);
    return;
}


int main(int argc, char * argv[])
{
    int *sum; // host sum
    int *dev_sum; // device sum
    int length;
    int *array;
    int i;
    int *array2;
    int *dev_array;
    int *dev_array2;


    srand(time(0));

    if(argc != 2)
    {
        std::cout << "usage:   ./addArray.elf <length of random array>\n";
        exit(0);
    }
    length = atoi(argv[1]);
    if(length > 65535)
    {
        std::cout << "please pick an array length less than 65535\n";
        exit(0);
    }

    if(length == 0)
    {
        std::cout<< "please specify a length > 0\n";
        exit(0);
    }
    hipMalloc( (void**)&dev_sum, sizeof(int) * length );
    hipMalloc( (void**)&dev_array, sizeof(int) * length );
    hipMalloc( (void**)&dev_array2, sizeof(int) * length );
    sum = (int*)malloc(sizeof(int) * length);
    memset((void*)sum, 0, sizeof(int) * length);
    
    generateArray(&array, length);
    generateArray(&array2, length);
    //debug(array, length);
    //debug(array2, length);
    hipMemcpy (dev_array, array, length * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy (dev_array2, array2, length * sizeof(int), hipMemcpyHostToDevice);

    addArray<<<length,1>>>( dev_array, dev_array2, dev_sum, length);

    hipMemcpy(sum, dev_sum, length * sizeof(int), hipMemcpyDeviceToHost);

    for(i = 0; i < length; i++)
        printf("%5d: %3d + %3d = %4d\n", i, array[i], array2[i], sum[i]);

    hipFree(dev_sum);
    hipFree(dev_array);
    hipFree(dev_array2);
    



    
    return 0;
}
    


    


