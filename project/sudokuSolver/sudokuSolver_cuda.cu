/*-------------------------------------------------------------------------------------------------------------
 *   Name:		    Jason Bisnette
 *   Class:		    CSC 718
 *   Professor:	    Dr. Yong Wang
 *   Assignment:	Final Project
 *   Due Date:	    08 December 2021
 *   File Name: 	sudokuSolver_cuda.cu
 *   Description:
 *              CUDA Version:
 *              Programming Assignment: A simple Sudoku puzzle solver for a puzzle size(s) of 9 x 9. 
 *              This program uses a backtracking algorithm to complete Sudoku puzzles: https://bit.ly/3d3kGTL
 *
 *   Notes:		- Puzzle files are included in the sudokuBoards folder of varying sizes.
 *              
 *              - Blank spaces are annotated with 0s in the puzzle file.
 *   
 *              Ex (9x9 puzzle):                            
 *              9                                           
 *              0 0 0 2 6 0 7 0 1                           
 *              6 8 0 0 7 0 0 9 0                           
 *              1 9 0 0 0 4 5 0 0                           
 *              8 2 0 1 0 0 0 4 0                           
 *              0 0 4 6 0 2 9 0 0                           
 *              0 5 0 0 0 3 0 2 8                           
 *              0 0 9 3 0 0 0 7 4                           
 *              0 4 0 0 5 0 0 3 6                           
 *              7 0 3 0 1 8 0 0 0                           
 *              
 *              For Sudoku, a candidate is valid if the following constraints are true:
 *              - Each row has unique numbers from 1 – size or empty spaces.
 *              - Each column has unique numbers from 1 – size or empty spaces.
 *              - Each sub-grid (minigrid) of 1 – size has the numbers 1 - size or empty spaces.
 * 
 *              Goal (only 1 to win):
 *              - Fill in the numbers from 1 - size exactly once in each row, column, and minigrid (sqrt(size))
 *               region.
 * 
 *              Backtracking termination conditions (Only 2):
 *              - The sudoku is already filled, meaning there is no white space.
 *              - There are no more empty spots left for the algorithm to check, and the current candidate
 *               does not reach the goal.
 *-----------------------------------------------------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>   // pow()
#include <string.h>
#include <hip/hip_runtime.h>
#include <time.h>

#include <cuda_device_runtime_api.h>

#ifndef __CUDACC__
    #define __CUDACC__
#endif

// 9x9 Sudoku
#define N 9
#define MINIGRID_SIZE 3 // sqrt(N)
#define CELLS 81
#define ITERATIONS 18 // number of ITERATIONS to run Searh

// 16x16 Sudoku
//#define N 16
//#define MINIGRID_SIZE 4
//#define CELLS 256

/*
 * Max puzzles for allowed for tree. Maximum size for int is 2^30 otherwise overflow
 * May need to be adjusted for harder (less filled in) puzzles
 * Works for > 16 (worst case for 9x9 81 cell sudoku)
 * Might need to change to long or long long for larger more complicated puzzles but this will require
 * altering the data types to other functions.
*/
const int MAX_PUZZLES = pow(2,28);

// Reads in puzzle file to the 1D array : sudoku.
void readPuzzle(char *FileName, int *sudoku) {
    FILE * file = fopen(FileName, "r");

    if (file == NULL) {
        printf("File load fail!\n"); return;
    }

    char trash; // Holds first line in file to be used later for doing larger puzzles
    char num;  // Holds the puzzle lines after every read

    // Throw this away for now ...
    if (!fscanf(file, "%c\n", &trash)) { 
        printf("File loading error!\n");
        exit(-1);
    }

    // Read puzzle from the file
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            if (!fscanf(file, "%c\n", &num)) {
                printf("File loading error!\n");
                exit(-1);
            }

            // Puts #s and 0s in the proper spot in array.
            if (num >= '1' && num <= 'N') { // If # > 0 put in array otherwise place the 0.
                // The idea here is the puzzle is stored in a 1D flat array instead of 2D. So to reference
                // row/column : row 1 is in element 0-8, row 2 is in element 9-17, etc...
                sudoku[i * N + j] = (int) (num - '0'); // removes null terminator and puts # in array
            } else { 
                sudoku[i * N + j] = 0; // Put a 0.
            }
        }
    }
} // end readPuzzle()

// Prints the puzzle to stdout
void printSudoku(int *sudoku) {
    for (int i = 0; i < N; i++) {
        if (i % MINIGRID_SIZE == 0) {
            printf("\n");
        }
        for (int j = 0; j < N; j++) {
            if (j % MINIGRID_SIZE == 0) {
            printf(" | ");
            }
            printf("%2d ", sudoku[i * N + j]);
        }
        printf(" |\n");
    }
    printf("\n\n");
} // end printSudoku()

// This function takes in a bool array and clears it to false.
__device__
void clearArray(bool *seenArray, int size) {
    
    for (int i = 0; i < size; i++) {
        seenArray[i] = false;
    }
} // end clearArray

 // Checks the puzzle to see if it is valid.
__device__
bool validBoard(const int *sudoku) {
    
    bool seen[N];           // Array of size N to track valid #s
    clearArray(seen, N);    // Clears array

    // Check rows. Rows are stored contiguously (1D array). To access row and col : sudoku[row * N + col]
    for (int row = 0; row < N; row++) {
        clearArray(seen, N);
        for (int col = 0; col < N; col++) {
            int valid = sudoku[row * N + col];
            if (valid != 0) {
                if (seen[valid - 1]) {
                    return false;
                } else {
                    seen[valid - 1] = true;
                }
            }
        }
    } // end row check

    // Check columns. Rows are stored contiguously (1D array). To access row and col : sudoku[row * N + col]
    for (int col = 0; col < N; col++) {
        clearArray(seen, N);

        for (int row = 0; row < N; row++) {
            int valid = sudoku[row * N + col];

            if (valid != 0) {
                if (seen[valid - 1]) {
                    return false;
                } else {
                    seen[valid - 1] = true;
                }
            }
        }
    } // end column check

    // Check minigrids
    for (int rowID = 0; rowID < MINIGRID_SIZE; rowID++) {
        for (int colID = 0; colID < MINIGRID_SIZE; colID++) {
            clearArray(seen, N);
            for (int row = 0; row < MINIGRID_SIZE; row++) {
                for (int col = 0; col < MINIGRID_SIZE; col++) {
                    int valid = sudoku[(rowID * MINIGRID_SIZE + row) * N + (colID * MINIGRID_SIZE + col)];
                    if (valid != 0) {
                        if (seen[valid - 1]) {
                            return false;
                        } else {
                            seen[valid-1] = true;
                        }
                    }
                }
            }
        }
    } // end minigrid check
    return true; // Valid board
} // end validBoard()

/**
 * changedIndex is between 0 and N * N - 1. Stores the index of the Sudoku that was changed
 * 
 * Assumes the Sudoku does not have the value at the index. 
 * 
 * Checks for validity for the new change.
  */
__device__
bool validBoard(const int *sudoku, int changedIndex) {

    int row = changedIndex / N;
    int col = changedIndex % N;

    // If index is less than 0, then default
    if (changedIndex < 0) {
        return validBoard(sudoku); // Call validboard(puzzle)
    }

    // Keep index between 1 and N (size of 1D of the sudoku)
    if ((sudoku[changedIndex] < 1) || (sudoku[changedIndex] > N)) {
        return false; // Not valid 
    }

    bool seen[N];           // Array for seen #s
    clearArray(seen, N);   // Create seenArray

    // Check rows. Rows are stored contiguously (1D array). To access row and col : sudoku[row * N + col]
    for (int i = 0; i < N; i++) {
        int valid = sudoku[row * N + i];
        if (valid != 0) {
            if (seen[valid - 1]) {
                return false;
            } else {
                seen[valid - 1] = true;
            }
        }
    } // end row check

    // Check columns. Rows are stored contiguously (1D array). To access row and col : sudoku[row * N + col]
    clearArray(seen, N);
    for (int j = 0; j < N; j++) {
        int valid = sudoku[j * N + col];
        if (valid != 0) {
            if (seen[valid - 1]) {
                return false;
            } else {
                seen[valid - 1] = true;
            }
        }
    } // end column check

    // Check minigrid. MINIGRID_SIZE is minigrid size or sqrt(N)
    int rowID = row / MINIGRID_SIZE; 
    int colID = col / MINIGRID_SIZE; 

    clearArray(seen, N); // Create seenArray

    // Moves within the minigrids
    for (int i = 0; i < MINIGRID_SIZE; i++) {
        for (int j = 0; j < MINIGRID_SIZE; j++) {
            int valid = sudoku[(rowID * MINIGRID_SIZE + i) * N + (colID * MINIGRID_SIZE + j)];
            if (valid != 0) {
                if (seen[valid - 1]) {
                    return false; // Not valid
                } else {
                    seen[valid - 1] = true;
                }
            }
        }
    } // end minigrid check
    return true; // Sudoku is valid
} // end validBoard() <- w/index

// Backtrack has each thread try to solve a different sudoku in the puzzle array.
__global__
void backtrack(int *puzzle,const int numBoards,int *emptySpaces,int *numEmptySpaces,int *finished,int *solvedSudoku) {

    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int *currentPuzzle;         // Location for current puzzle being worked on
    int *currentEmptySpaces;    // Tracks location of empty spaces in currenly loaded puzzle
    int currentNumEmptySpaces;  // Tracks # of empty spaces in current puzzle

    // While more work to do.
    // Finished var is key to stopping the kernel so it does not iterate until end.
    while ((*finished == 0) && (index < numBoards)) {
    
        int emptyIndex = 0; // Set the emptyIndex to 0

        currentPuzzle = puzzle + index * CELLS;             // Gets current puzzle
        currentEmptySpaces = emptySpaces + index * CELLS;   // Gets current empty spaces
        // Each value stores the number of empty spaces in the corresponding sudoku.
        currentNumEmptySpaces = numEmptySpaces[index];      // Gets # of empty spaces for current puzzle

        // While there are still spots in the Sudoku that are empty do this
        while ((emptyIndex >= 0) && (emptyIndex < currentNumEmptySpaces)) {
            
            currentPuzzle[currentEmptySpaces[emptyIndex]]++; // Move to next empty spot

            // If not a valid board and all the #s are attemped just backtrack or just continue to next iteration
            if (!validBoard(currentPuzzle, currentEmptySpaces[emptyIndex])) {
                if (currentPuzzle[currentEmptySpaces[emptyIndex]] >= N) {
                    currentPuzzle[currentEmptySpaces[emptyIndex]] = 0;
                    emptyIndex--; // Backup
                }
            }
            else { // If the Sudoku is valid increment the emptyIndex and continue
                emptyIndex++; // Move forward
            }
        } 

        // Solution found, stop kernel and copy the results to the solveSodoku array
        if (emptyIndex == currentNumEmptySpaces) {
            *finished = 1; // <-- this stops the kernel
            for (int i = 0; i < N * N; i++) {
                solvedSudoku[i] = currentPuzzle[i]; // Copy current solved puzzle to solvedSudoku array
            }
        }
        index += gridDim.x * blockDim.x;
    }
} // end backtrack()

// Call backtrack kernel
void deviceBacktrack(const  int blocks,const  int threadsPerBlock,int *puzzle,const int numBoards, int *emptySpaces, int *numEmptySpaces, int *finished, int *solvedSudoku) {

    backtrack<<<blocks, threadsPerBlock>>>(puzzle, numBoards, emptySpaces, numEmptySpaces, finished, solvedSudoku);
} // end deviceBacktrack()


// This kernel takes old puzzles and finds all possible puzzles by filling in the empty spaces.
// puzzleIndex must start at zero!
// This is a breadth first search as it was easier than depth first to parallelize!
__global__
void deviceSearch(int *oldPuzzles,int *newPuzzles,int puzzleIndexReset,int *puzzleIndex,int *emptySpaces,int *emptyCount) {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    
    while (index < puzzleIndexReset) {
    
        int found = 0; // Set found to 0 to enter for loop

        // Find next empty position
        for (int i = (index * N * N); (i < (index * N * N) + N * N) && (found == 0); i++) {
            if (oldPuzzles[i] == 0) {               // Empty found!
                found = 1;                          // Set to 1 and if not found set to 0
                int location = (i - N * N * index);   // Set location in array
                int row = (location / N);             // Get proper row
                int col = (location % N);             // Get proper column
                
                // Find correct # to fit in location
                for (int num = 1; num <= N; num++) {
                    
                    int fits = 1; // Set to fits and if num does not fit change to 0

                    // Check rows
                    for (int numRow = 0; numRow < N; numRow++) { 
                        if (oldPuzzles[numRow * N + col + N * N * index] == num) {
                            fits = 0;
                        }
                    }
                    
                    // Check columns
                    for (int numCol = 0; numCol < N; numCol++) {  
                        if (oldPuzzles[row * N + numCol + N * N * index] == num) {
                            fits = 0; // Does not fit
                        }
                    }

                    // Check minigrids
                    for (int numRow = MINIGRID_SIZE * (row / MINIGRID_SIZE); numRow < MINIGRID_SIZE; numRow++) {
                        for (int numCol = MINIGRID_SIZE * (col / MINIGRID_SIZE); numCol < MINIGRID_SIZE; numCol++) {
                            if (oldPuzzles[numRow * N + numCol + N * N * index] == num) {
                                fits = 0; // Does not fit
                            }
                        }
                    }

                    // If it made it this far num fits by default
                    if (fits == 1) {
                        
                        // Move to the next Puzzle (prevent race condition)
                        int nextPuzzleIndex = atomicAdd(puzzleIndex, 1); 
                        int emptiesIndex = 0;  // Reset emptiesIndex

                        // Copy puzzles, old -> new
                        for (int numRow = 0; numRow < N; numRow++) {
                            for (int numCol = 0; numCol < N; numCol++) {
                                newPuzzles[nextPuzzleIndex * CELLS + numRow * N + numCol] = oldPuzzles[index * CELLS + numRow * N + numCol];
                                if (oldPuzzles[index * CELLS + numRow * N + numCol] == 0 && (numRow != row || numCol != col)) {

                                    emptySpaces[emptiesIndex + CELLS * nextPuzzleIndex] = numRow * N + numCol;
                                    emptiesIndex++; // Move to the next spot
                                }
                            }
                        }
                        // Set empties count/index for new puzzle
                        emptyCount[nextPuzzleIndex] = emptiesIndex;
                        // Put number in puzzle at proper location
                        newPuzzles[nextPuzzleIndex * CELLS + row * N + col] = num; 
                    }
                }
            }
        }
        index += blockDim.x * gridDim.x;
    }
} // end deviceSearch()


void callSearchKernel(const int blocks,const int threadsPerBlock,int *oldPuzzles,int *newPuzzles,int puzzleIndexReset,int *puzzleIndex,int *emptySpaces,int *emptyCount) {
    
    deviceSearch<<<blocks, threadsPerBlock>>>(oldPuzzles, newPuzzles, puzzleIndexReset, puzzleIndex, emptySpaces, emptyCount);
} // end callSearchKernel()

int main(int argc, char *argv[]) {
    
    if (argc != 4){
        printf("\n\tERROR: Incorrect # of args.\n"
        "\t\t\tSyntax: ./sudokuSolver <threads per block> <max number of blocks> <puzzleFile>\n\n");
        exit(-1);
    }

    clock_t time;               // clock struct from <time.h>
    double timeTaken;           // End time variable
    int *sudoku;                // Sudoku puzzle 1D array
    /*
     *  newPuzzles and oldPuzzles contain multiple puzzles in a 1D array. To access a specific puzzle within 
     *  the array use newPuzzle[board# * N * N + row * N + column]
    */
    int *newPuzzles;            // New puzzles after search
    int *oldPuzzles;            // The previous puzzle
    int *emptySpaces;           // Location of empty spaces
    int *emptyCount;            // Number of empty spaces
    int *puzzleIndex;           // Location of new puzzle
    int puzzleIndexReset = 1;   // Need to set puzzleIndex to zero every iteration so use this to track them
    int *deviceFinished;        // Bool var for solution found
    int *deviceSolved;          // To store solvedSudoku Sudoku from device
    int *solvedSudoku ;         // Stores solved Sudoku
    int threadsPerBlock = atoi(argv[1]); // Holds user input # threads from arg and convert to int
    int maxBlocks = atoi(argv[2]);       // Holds user input maxblocks from arg and convert to int
    char* filename = argv[3];   // Filename of the starting sudoku
    int puzzleCount;             // number of puzzle after a call to search function

    time = clock(); // Start timer

    // Allocate space for puzzle
    if((sudoku = (int*)malloc((N * N) * sizeof(int))) == NULL) { 
        printf("Error: Not enough memory.\n");
        exit(-1); // Exit/error cannot aquire memory
    }

    readPuzzle(filename, sudoku); // Read in sudoku from file

    // Allocate space for solved puzzle
    if((solvedSudoku = (int*)malloc((N * N) * sizeof(int))) == NULL) { 
        printf("Error: Not enough memory.\n");
        exit(-1); // Exit/error cannot aquire memory
    }

    memset(solvedSudoku, 0, (N * N) * sizeof(int)); // Fill solvedSudoku array with 0s

    printf("\nPuzzle file -> %s is : %d x %d\n\n", argv[3], N, N); // Puzzle name and size of puzzle 
    printf("Loaded Sudoku Puzzle: \n");
    printSudoku(sudoku); // Print sudoku from file before solution

    hipMalloc(&emptySpaces, MAX_PUZZLES * sizeof(int));                 // Allocate memory
    hipMalloc(&emptyCount, (MAX_PUZZLES / CELLS + 1) * sizeof(int));    // Allocate memory
    hipMalloc(&newPuzzles, MAX_PUZZLES * sizeof(int));                  // Allocate memory
    hipMemset(newPuzzles, 0, MAX_PUZZLES * sizeof(int));                // Fill with 0s
    hipMalloc(&oldPuzzles, MAX_PUZZLES * sizeof(int));                  // Allocate memory
    hipMemset(oldPuzzles, 0, MAX_PUZZLES * sizeof(int));                // Fill with 0s
    // copy the sudoku to oldPuzzles on device. (cudaMemcpyHostToDevice = 1)
    hipMemcpy(oldPuzzles, sudoku, N * N * sizeof(int), hipMemcpyHostToDevice); 
    hipMalloc(&puzzleIndex, sizeof(int));                       // Allocate memory
    hipMemset(puzzleIndex, 0, sizeof(int));                     // Make puzzleIndex 0

    // Call the kernel to generate puzzle
    callSearchKernel(maxBlocks,threadsPerBlock,oldPuzzles,newPuzzles,puzzleIndexReset,puzzleIndex,emptySpaces,emptyCount);

    // Loop through ITERATIONS to generate more puzzles in the binary tree
    for (int i = 0; i < ITERATIONS; i++) {

        // Copy puzzleIndex to puzzleCount 
        hipMemcpy(&puzzleCount, puzzleIndex, sizeof(int), hipMemcpyDeviceToHost);
        hipMemset(puzzleIndex, 0, sizeof(int)); // Change puzzleIndex to 0

        // Left
        if (i % 2 == 0) {
            callSearchKernel(maxBlocks,threadsPerBlock,newPuzzles,oldPuzzles,puzzleCount,puzzleIndex,emptySpaces,emptyCount);
        }
        // Right
        else {
            callSearchKernel(maxBlocks,threadsPerBlock,oldPuzzles,newPuzzles,puzzleCount,puzzleIndex,emptySpaces,emptyCount);
        }
    }

    // Copy puzzleIndex from device to puzzleCount on host
    hipMemcpy(&puzzleCount, puzzleIndex, sizeof(int), hipMemcpyDeviceToHost);
    hipMalloc(&deviceFinished, sizeof(int));           // Allocate memory
    hipMemset(deviceFinished, 0, sizeof(int));         // Fill with 0s
    hipMalloc(&deviceSolved, (N * N) * sizeof(int));   // Allocate memory
    // Copy sudoku from host to deviceSolved on device
    hipMemcpy(deviceSolved, sudoku, (N * N) * sizeof(int), hipMemcpyHostToDevice);

    // If odd # of ITERATIONS then send to oldPuzzle instead of newPuzzle
    if (ITERATIONS % 2 == 1) {
        newPuzzles = oldPuzzles;
    }

    // Call backtracking algorithm
    deviceBacktrack(maxBlocks,threadsPerBlock,newPuzzles,puzzleCount,emptySpaces,emptyCount,deviceFinished,deviceSolved);

    // Puzzle soved! Copy deviceSolved from device to solvedSudoku on host
    hipMemcpy(solvedSudoku, deviceSolved, (N * N) * sizeof(int), hipMemcpyDeviceToHost);

    time = clock() - time;                      // Stop timer
    timeTaken = ((double)time)/CLOCKS_PER_SEC;  // Calculate time

    // Print results
    printf("\nPuzzle Solved: \n");
    printSudoku(solvedSudoku);
    printf("\t\tThe %d x %d puzzle %s was solved in %f seconds\n\n", N, N, argv[3], timeTaken);

    // Free all allocated memory
    free(sudoku);
    free(solvedSudoku);
    hipFree(emptySpaces);
    hipFree(emptyCount);
    hipFree(newPuzzles);
    hipFree(oldPuzzles);
    hipFree(puzzleIndex);
    hipFree(deviceFinished);
    hipFree(deviceSolved);

    return 0; 
} // end main()